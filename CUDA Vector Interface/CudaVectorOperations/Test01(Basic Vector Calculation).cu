#include "hip/hip_runtime.h"
//Basic Demonstration of how to use Kernal Vector Operations
#include <iostream>
#include "CudaVectorOperations.cuh"

__global__ void addKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
}

__global__ void multiplyKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
    printf("%d * %d = %d\n", a[i], b[i], c[i]);
}

int main(void) {
    KernalVectorOperation kvOPAdd = *addKernel;
    KernalVectorOperation kvOPMult = *multiplyKernel;
    int* output = new int[50000];
    int* a = new int[50000];
    int* b = new int[50000];

    for (int i = 1; i <= 50000; i++) {
        a[i - 1] = i;
        b[i - 1] = i * 10;
    }

    performVectorOperation(output, a, b, 50000, kvOPAdd);

    performVectorOperation(output, a, b, 50000, kvOPMult);

    delete[] output;
    delete[] a;
    delete[] b;

    return 0;

}