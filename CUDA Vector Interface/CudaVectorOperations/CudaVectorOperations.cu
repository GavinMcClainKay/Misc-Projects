#include <iostream>
#include "CudaVectorOperations.cuh"

hipError_t setGPU(int device) {
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    return cudaStatus;
}

hipError_t allocateMemory(void** devPTR, unsigned int size) {
    hipError_t cudaStatus = hipMalloc((void**)&devPTR, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    return cudaStatus;
}

hipError_t performVectorOperation(int* output, const int* in_a, const int* in_b, unsigned int size, KernalVectorOperation operation) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipError_t cudaStatus;
    
    cudaStatus = setGPU(0);

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = allocateMemory((void**)&dev_a, size * sizeof(int));
    cudaStatus = allocateMemory((void**)&dev_b, size * sizeof(int));
    cudaStatus = allocateMemory((void**)&dev_c, size * sizeof(int));



    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, in_a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, in_b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.
    operation<<<50, 1000>>>(dev_c, dev_a, dev_b);



    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
